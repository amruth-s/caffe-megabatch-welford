#include "hip/hip_runtime.h"
#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>

#include <cmath>

#include "caffe/common.hpp"
#include "caffe/util/math_functions.hpp"

#include "caffe/util/math_functions.hpp"
#define C1_KERNEL_NUM_BLOCKS 2048
#define C2_KERNEL_NUM_BLOCKS 512
#define C3_KERNEL_NUM_BLOCKS 256
#define C1_INPUT_COLS_PER_BLOCK 4
#define C2_C3_INPUT_VECS_PER_BLOCK 4
#define CUDA_WARP_SIZE 32
#define NUM_NEURONS_PER_BLOCK 16

namespace caffe {

//this is the kerbel where one warp handles computation of one neuron
//TODO: low level optimizations are required!
__global__ void my_conv_kernel_generic_f(const float* matrix1, const float* matrix2, float* outputs, const int M, const int N, const int K)
{
  const unsigned int tid = threadIdx.x;
  const unsigned int bid = blockIdx.x;
  float partial_sum = 0.0;
  const unsigned int warp_id = tid >> 5;// / CUDA_WARP_SIZE;
  const unsigned int intra_warp_tid = tid & 31;// % CUDA_WARP_SIZE;
  const int g_warp_id=(bid<<4)+warp_id;
  const unsigned int matrix2_col_id=g_warp_id%N; //global warp id
  const unsigned int matrix1_row_id = g_warp_id/N;
  int matrix2_index = (matrix2_col_id*K) + intra_warp_tid;
  int matrix1_index = (matrix1_row_id *K)+intra_warp_tid;
  if (matrix2_index>= K*N || matrix1_index >=K*M)
          return;
  partial_sum += matrix1[matrix1_index] * matrix2[matrix2_index];
  #pragma unroll
  for (int q=32;intra_warp_tid+q<K;q+=32) {
      matrix2_index = 32 + matrix2_index;
      matrix1_index = matrix1_index + 32;
      if (matrix2_index >= K*N  || matrix1_index >=K*M)
            return;
      partial_sum += matrix1[matrix1_index] * matrix2[matrix2_index];
  }
    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 16, 32);
    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 8, 32);
    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 4, 32);
    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 2, 32);
    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 1, 32);
    if(intra_warp_tid == 0)
          outputs[g_warp_id] = partial_sum;
}

__global__ void my_conv_kernel_generic_pruning(const float* matrix1, const float* matrix2, float* outputs, const int M, const int N, const int K, float *pruning, bool is_normal)
{
  float prun=pruning[(blockIdx.x<<4)+(threadIdx.x >> 5)];
  const unsigned int tid = threadIdx.x;
  const unsigned int bid = blockIdx.x;
  float partial_sum = 0.0;
  const unsigned int warp_id = tid >> 5;// / CUDA_WARP_SIZE;
  const unsigned int intra_warp_tid = tid & 31;// % CUDA_WARP_SIZE;
  const int g_warp_id=(bid<<4)+warp_id;
  if(prun==1234){
	  const unsigned int matrix2_col_id=g_warp_id%N; //global warp id
	  const unsigned int matrix1_row_id = g_warp_id/N;
	  int matrix2_index = (matrix2_col_id*K) + intra_warp_tid;
	  int matrix1_index = (matrix1_row_id *K)+intra_warp_tid;
	  if (matrix2_index>= K*N || matrix1_index >=K*M)
	          return;
	  partial_sum += matrix1[matrix1_index] * matrix2[matrix2_index];
	  #pragma unroll
	  for (int q=32;intra_warp_tid+q<K;q+=32) {
	      matrix2_index = 32 + matrix2_index;
	      matrix1_index = matrix1_index + 32;
	      if (matrix2_index >= K*N  || matrix1_index >=K*M)
	            return;
	      partial_sum += matrix1[matrix1_index] * matrix2[matrix2_index];
	  }
	    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 16, 32);
	    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 8, 32);
	    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 4, 32);
	    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 2, 32);
	    partial_sum += __shfl_down_sync(0xffffffff, partial_sum, 1, 32);
	    if(intra_warp_tid == 0)
	          outputs[g_warp_id] = partial_sum;
  } else {
	    if(intra_warp_tid == 0)
		  outputs[g_warp_id] = prun;
  }
}

template<>
void test_conv_function<float>(const float* weights, const float* inputs, float* outputs, int M, int N, int K, float* pruning, const bool is_normal) {
	  if(is_normal)
	    my_conv_kernel_generic_f<<<((M*N)>>4), CAFFE_CUDA_NUM_THREADS>>>(weights, inputs, outputs, M, N, K);
	  else
	    my_conv_kernel_generic_pruning<<<((M*N)>>4), CAFFE_CUDA_NUM_THREADS>>>(weights, inputs, outputs, M, N, K, pruning,is_normal);
}

template<>
void test_conv_function<double>(const double* weights, const double* inputs, double* outputs, int M, int N, int K, float* pruning, bool is_normal) {
}

template <>
void caffe_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const float alpha, const float* A, const float* B, const float beta,
    float* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasSgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
    const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
    const double alpha, const double* A, const double* B, const double beta,
    double* C) {
  // Note that cublas follows fortran order.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_CHECK(hipblasDgemm(Caffe::cublas_handle(), cuTransB, cuTransA,
      N, M, K, &alpha, B, ldb, A, lda, &beta, C, N));
}

template <>
void caffe_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const float alpha, const float* A, const float* x,
    const float beta, float* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasSgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA, const int M,
    const int N, const double alpha, const double* A, const double* x,
    const double beta, double* y) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_CHECK(hipblasDgemv(Caffe::cublas_handle(), cuTransA, N, M, &alpha,
      A, N, x, 1, &beta, y, 1));
}

template <>
void caffe_gpu_axpy<float>(const int N, const float alpha, const float* X,
    float* Y) {
  CUBLAS_CHECK(hipblasSaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void caffe_gpu_axpy<double>(const int N, const double alpha, const double* X,
    double* Y) {
  CUBLAS_CHECK(hipblasDaxpy(Caffe::cublas_handle(), N, &alpha, X, 1, Y, 1));
}

void caffe_gpu_memcpy(const size_t N, const void* X, void* Y) {
  if (X != Y) {
    CUDA_CHECK(hipMemcpy(Y, X, N, hipMemcpyDefault));  // NOLINT(caffe/alt_fn)
  }
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float *X) {
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double *X) {
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
}

template <>
void caffe_gpu_scal<float>(const int N, const float alpha, float* X,
                           hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_scal<double>(const int N, const double alpha, double* X,
                            hipStream_t str) {
  hipStream_t initial_stream;
  CUBLAS_CHECK(hipblasGetStream(Caffe::cublas_handle(), &initial_stream));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), str));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), N, &alpha, X, 1));
  CUBLAS_CHECK(hipblasSetStream(Caffe::cublas_handle(), initial_stream));
}

template <>
void caffe_gpu_axpby<float>(const int N, const float alpha, const float* X,
    const float beta, float* Y) {
  caffe_gpu_scal<float>(N, beta, Y);
  caffe_gpu_axpy<float>(N, alpha, X, Y);
}

template <>
void caffe_gpu_axpby<double>(const int N, const double alpha, const double* X,
    const double beta, double* Y) {
  caffe_gpu_scal<double>(N, beta, Y);
  caffe_gpu_axpy<double>(N, alpha, X, Y);
}

template <>
void caffe_gpu_dot<float>(const int n, const float* x, const float* y,
    float* out) {
  CUBLAS_CHECK(hipblasSdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_dot<double>(const int n, const double* x, const double* y,
    double * out) {
  CUBLAS_CHECK(hipblasDdot(Caffe::cublas_handle(), n, x, 1, y, 1, out));
}

template <>
void caffe_gpu_asum<float>(const int n, const float* x, float* y) {
  CUBLAS_CHECK(hipblasSasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_asum<double>(const int n, const double* x, double* y) {
  CUBLAS_CHECK(hipblasDasum(Caffe::cublas_handle(), n, x, 1, y));
}

template <>
void caffe_gpu_scale<float>(const int n, const float alpha, const float *x,
                            float* y) {
  CUBLAS_CHECK(hipblasScopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasSscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <>
void caffe_gpu_scale<double>(const int n, const double alpha, const double *x,
                             double* y) {
  CUBLAS_CHECK(hipblasDcopy(Caffe::cublas_handle(), n, x, 1, y, 1));
  CUBLAS_CHECK(hipblasDscal(Caffe::cublas_handle(), n, &alpha, y, 1));
}

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void caffe_gpu_set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));  // NOLINT(caffe/alt_fn)
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template void caffe_gpu_set<int>(const int N, const int alpha, int* Y);
template void caffe_gpu_set<float>(const int N, const float alpha, float* Y);
template void caffe_gpu_set<double>(const int N, const double alpha, double* Y);

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <>
void caffe_gpu_add_scalar(const int N, const float alpha, float* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <>
void caffe_gpu_add_scalar(const int N, const double alpha, double* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <>
void caffe_gpu_add<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_add<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <>
void caffe_gpu_sub<float>(const int N, const float* a, const float* b,
    float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_sub<double>(const int N, const double* a, const double* b,
    double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <>
void caffe_gpu_mul<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_mul<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a,
    const Dtype* b, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <>
void caffe_gpu_div<float>(const int N, const float* a,
    const float* b, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <>
void caffe_gpu_div<double>(const int N, const double* a,
    const double* b, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void abs_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = abs(a[index]);
  }
}

template <>
void caffe_gpu_abs<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_abs<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  abs_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}


template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <>
void caffe_gpu_exp<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_exp<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  exp_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void log_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = log(a[index]);
  }
}

template <>
void caffe_gpu_log<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_log<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  log_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a,
    const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <>
void caffe_gpu_powx<float>(const int N, const float* a,
    const float alpha, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <>
void caffe_gpu_powx<double>(const int N, const double* a,
    const double alpha, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sqrt_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = sqrt(a[index]);
  }
}

template <>
void caffe_gpu_sqrt<float>(const int N, const float* a, float* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

template <>
void caffe_gpu_sqrt<double>(const int N, const double* a, double* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sqrt_kernel<double><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, y);
}

DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sign, y[index] = (Dtype(0) < x[index])
                                      - (x[index] < Dtype(0)));
DEFINE_AND_INSTANTIATE_GPU_UNARY_FUNC(sgnbit, y[index] = signbit(x[index]));

void caffe_gpu_rng_uniform(const int n, unsigned int* r) {
  CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), r, n));
}

template <>
void caffe_gpu_rng_uniform<float>(const int n, const float a, const float b,
                                  float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_uniform<double>(const int n, const double a, const double b,
                                   double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    caffe_gpu_scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    caffe_gpu_add_scalar(n, a, r);
  }
}

template <>
void caffe_gpu_rng_gaussian(const int n, const float mu, const float sigma,
                            float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void caffe_gpu_rng_gaussian(const int n, const double mu, const double sigma,
                            double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}

}  // namespace caffe
