#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/conv_layer.hpp"

namespace caffe {

vector <float *> pruning1; //used for entry matrix calculation
vector <float *> pruning2;
int epoch=0; //counts no of epochs
int images=0; //counts no of images fed to neural net

template <typename Dtype>
__global__ void Threshold_pruning(const int n,
    Dtype* in, float* out) {
  CUDA_KERNEL_LOOP(index, n) {
	  if (in[index]>10) //threshold=10
		  out[index] =1234; //some magic number for me to identify if a neuron is marked for skipping computation or not based on threshold value.
	  else
		  out[index] = in[index];
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const int p_t =0;  //pre-stan interval
  const int Z=1; //skipping interval
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int count = this->top_dim_;
    if (epoch==0 && this->phase_ == TRAIN){ //repeat this until we allocate memory for entry matrix for all neurons for all images in dataset.
	    float* d_x = NULL;
	    hipMalloc(&d_x, sizeof(float)*count*this->num_);
	    hipMemset(d_x,0,count*this->num_*sizeof(float));
	    if (count == 11520) //output dimension of LeNet are 11520 for 1st layer and 3200 for 2nd layer.
	         pruning1.push_back(d_x);
	    else
        	 pruning2.push_back(d_x);
    }
    for (int n = 0; n < this->num_; ++n) {
      if (this->phase_ == TEST || ((this->phase_ == TRAIN) && (epoch<p_t))){
          this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
              top_data + n * this->top_dim_, NULL,true);
      } else {
	  if ((epoch-p_t)% (Z+1)==0){
		this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
	              top_data + n * this->top_dim_,NULL,true);
		if (count == 11520)
			 Threshold_pruning<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data + n * this->top_dim_, pruning1[images/100]+n * this->top_dim_);
		else 
			 Threshold_pruning<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, top_data + n * this->top_dim_, pruning2[images/100]+n * this->top_dim_);
		CUDA_POST_KERNEL_CHECK;
	  } else {
		if (count == 11520)
			this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight, top_data + n * this->top_dim_, pruning1[images/100]+n * this->top_dim_, false);
		else 
			this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight, top_data + n * this->top_dim_, pruning2[images/100]+n * this->top_dim_, false); 
	  }
      }
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
    if (this->phase_ == TRAIN){
	if(this->top_dim_ == 3200) //dimension of last layer
		images+=this->num_;
	if (images >=60000){ //MNIST has 60k images in total
		epoch++;
		images=0;
	}
    }
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff + n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data + n * this->bottom_dim_,
              top_diff + n * this->top_dim_, weight_diff);
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          this->backward_gpu_gemm(top_diff + n * this->top_dim_, weight,
              bottom_diff + n * this->bottom_dim_);
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
